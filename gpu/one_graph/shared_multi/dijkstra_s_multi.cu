#include "hip/hip_runtime.h"
//***************************************************************************
//  Broday Walker
//  Dr. Eduardo Colmenares
//   
//  Turing
//  Note: -rdc=true must be used as CUDA dynamic parallelism requires separate 
//  compilation and linking
//  Compilation:  /opt/cuda-9.0/bin/nvcc -arch=sm_37 -rdc=true dijkstra_s_multi.cu -o dijkstra_s_multi.exe
//  Execution: ./dijkstra_s_multi.exe < ~/5433/dijkstra/graphs/t_4096.txt > output4096.txt
//
//  Maverick2 GTX queue (use submission script)
//  Compilation: nvcc dijkstra_s_multi.cu -o dijkstra_s_multi.exe -std=c++11 -O3 -arch=compute_61 -code=sm_61 -rdc=true
//
//  Maverick2 V100 queue
//  Compilation: nvcc dijkstra_s_multi.cu -o dijkstra_s_multi.exe -std=c++11 -O3 -arch=compute_70 -code=sm_70 -rdc=true
//***************************************************************************

#include <iostream>
#include <fstream>
#include <vector>
#include <stack>
#include <climits>
#include <stdint.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

const int BLOCK_SIZE_X = 1024; // Number of threads per block on the x-axis

__global__ void relax(int *dist_d, int *parent_d, int *visited_d, int *adjMat_d, int u, int width)
{
    // Must use extern as the size of adjRow_ds is set by parent kernel
    extern __shared__ int adjRow_ds[];
    
    int tid_x = threadIdx.x;
    int b_id = blockIdx.x;
    int b_dim_x = blockDim.x;

    // u * width gets us to the row we want
    // blockIDx.x * blockDim.x + threadIdx.x gets our offset within the row
    int offset = u * width + (b_id * b_dim_x + tid_x);

    // This is used to reference dist_d, parent_d, and visited_d
    // This maps a thread to a vertex
    int t_offset = b_id * b_dim_x + tid_x;

    // Bring this block's portion of the adjacency row data for vertex u 
    // from global memory to shared memory
    adjRow_ds[tid_x] = adjMat_d[offset];
    __syncthreads();

    // Each thread will attempt to relax an adjacent vertex if an edge exists
    // If the vertex is unvisited and the edge exists
    // Self-loops are ignored
    if(t_offset != u && visited_d[t_offset] == 0 && adjRow_ds[tid_x] >= 0)
    {
        int v_dist = adjRow_ds[tid_x];

        if(dist_d[u] + v_dist < dist_d[t_offset])
        {
            // Relax  
            dist_d[t_offset] = dist_d[u] + v_dist;
            // Set parent
            parent_d[t_offset] = u;
        }
    }
}

__global__ void dijkstra(int *dist_d, int *parent_d, int *visited_d, int *adjMat_d, int s, int width)
{   
    int blocks;
    int tid_x = threadIdx.x;
    int shared_mem_size;
    bool all_visited = false;

    // Determine the number of blocks needed
    // For the purpose of this project, width (number of vertices) and BLOCK_SIZE_X will always
    // be a power of 2. Otherwise, I would take the ceiling of width / BLOCK_SIZE_X and check for
    // out-of-bounds threads in the child kernel.
    blocks = width / BLOCK_SIZE_X;

    shared_mem_size = (width / blocks) * sizeof(int);

    dim3 childGrid(blocks, 1);
    dim3 childBlock(BLOCK_SIZE_X, 1);

    if(tid_x == 0)
    {
        while(!all_visited)
        {
            all_visited = true;
            bool next_found = false;

            // These are u sed as the variables for the next vertex's index and minimum cost
            int u, u_min, index = 0;

            // Find the next unvisited vertex (if there is one)
            while (!next_found && index < width)
            {
                if(visited_d[index] == 0)
                {
                    u = index;
                    u_min = dist_d[index];
                    all_visited = false;
                    next_found = true;
                }

                index++;
            }

            if(!all_visited)
            {
                // Find shortest distance of all the unvisited vertices
                // Start at the first unvisited vertex encountered in the previous loop to 
                // try to save some computation.
                int i = u + 1;
                while(i < width)
                {
                    if(visited_d[i] == 0 && dist_d[i] < u_min)
                    {
                        u = i;
                        u_min = dist_d[i];
                    }

                    i++;
                }

                // Now we have the next vertex to process in u
                // Set u as visited
                visited_d[u] = 1;

                // Attempt to relax all vertices adjacent to vertex u
                relax<<<childGrid, childBlock, shared_mem_size>>>(dist_d, parent_d, visited_d, adjMat_d, u, width);
                hipDeviceSynchronize();
            }
        }
    }
}

// A kernel for testing if the adjacency matrix was actually copied to the constant
// memory on the device
__global__ void printAdjMat(int *test, int *adjMat_d, int width)
{
    int tid_x = threadIdx.x;
    
    for(int i = 0; i < width; i++)
        test[i * width + tid_x] = adjMat_d[i * width + tid_x];
}

int main()
{
    int vertices, cases = 1;

    // Declare the timer
    // Reference: 
    // https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    hipEvent_t begin, stop;
    hipEventCreate(&begin);
    hipEventCreate(&stop);
    float milliseconds = 0;

	ofstream outfile;
	outfile.open("timings_GPU_1B_global.txt");

    cin >> vertices;

    while(vertices != 0)
    {
        // Host Declarations
        int adj_size, dist_size, parent_size, visited_size, start, end, p;
        int *adjMat, *dist, *parent, *visited;
        stack<int> path;

        // Device declarations
        int *adjMat_d, *dist_d, *parent_d, *visited_d;

        // This is a linearized adjacency matrix
        adjMat = new int[vertices * vertices];
        dist = new int[vertices];
        parent = new int[vertices];
        visited = new int[vertices];

        // Find size of arrays in bytes
        adj_size = vertices * vertices * sizeof(int);
        dist_size = vertices * sizeof(int); // Equal to the number of vertices
        parent_size = vertices * sizeof(int);
        visited_size = vertices * sizeof(int);

        // Fill the adjacency-matrix with 0s
        for(int i = 0; i < vertices * vertices; i++)
            adjMat[i] = -1;

        // A vertex does not have a parent if its value is -1 (after running 
        // Dijkstra's algorithm, this will only be true for the starting vertex).
        for(int i = 0; i < vertices; i++)
        {
            dist[i] = INT_MAX;
            parent[i] = -1;
            visited[i] = 0;
        }

        // Fill the adjacency matrix with data
        for(int i = 0; i < vertices; i++)
        {
            // Temporary storage for adjacent vertices and the weight of the edge
            int num_edges, u, w;

            // Read in the number of adjacent vertices for the ith vertex
            cin >> num_edges;

            // Add the adjacent vertices to the linearized adjacency-matrix for the ith vertex
            for(int j = 0; j < num_edges; j++)
            {
                cin >> u >> w;
                int offset = i * vertices + u;

                if(adjMat[offset] == -1 || w < adjMat[offset])
                    // This is basically blockIdx.x * blockDim.x + threadIdx.x where blockIdx.x 
                    // corresponds with the iteration of the loop we are on
                    adjMat[offset] = w;
            }
        }

        cin >> start >> end;

        /* Start Dijkstra */

        // Set distance of source vertex to 0;
        dist[start] = 0;
              
        // Set the dimensions of the grid and blocks
        dim3 gridDim(1, 1);
        dim3 blockDim(1, 1);

        // Allocate memory on the device 
        hipMalloc((void **)&dist_d, dist_size);
        hipMemcpy(dist_d, dist, dist_size, hipMemcpyHostToDevice);

        hipMalloc((void **)&parent_d, parent_size);
        hipMemcpy(parent_d, parent, parent_size, hipMemcpyHostToDevice);

        hipMalloc((void **)&visited_d, visited_size);
        hipMemcpy(visited_d, visited, visited_size, hipMemcpyHostToDevice);

        hipMalloc((void **)&adjMat_d, adj_size);
        hipMemcpy(adjMat_d, adjMat, adj_size, hipMemcpyHostToDevice);

        /* Record start time */
        hipEventRecord(begin);

        // Invoke the kernel
        dijkstra<<<gridDim, blockDim>>>(dist_d, parent_d, visited_d, adjMat_d, start, vertices);

        /* Record end time */
        hipEventRecord(stop);

        // Copy the results back
        hipMemcpy(dist, dist_d, dist_size, hipMemcpyDeviceToHost);
        hipMemcpy(parent, parent_d, parent_size, hipMemcpyDeviceToHost);

        /* Block CPU execution until the specified event is recorded */
        hipEventSynchronize(stop);
        /* Returns the elapsed time in milliseconds to the first argument */
        hipEventElapsedTime(&milliseconds, begin, stop);

        // Free the device memory
        hipFree(dist_d);
        hipFree(parent_d);
        hipFree(visited_d);

        // Start at the end vertex and work back through the parent vertices to build the path
        p = end;
		path.push(p);
		while (p != start)
		{
			p = parent[p];
			path.push(p);
		}

		// Print result
		cout << "Case " << cases << ": Path =";

		while (!path.empty())
		{
            // NOTE
            // Add one to the vertex ID number if comparing results to UVA problem 341 
			cout << " " << path.top() + 1;
			path.pop();
		}

		cout << "; " << dist[end] << " second delay\n";


        // Free the host memory
        delete [] adjMat;
        delete [] dist;
        delete [] parent;
        delete [] visited;


        cases++;
        cin >> vertices;

    }

	// Print elapsed time
    outfile << "Elapsed time in milliseconds: " << milliseconds << '\n';

    outfile.close();

    return 0;
}